#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/driver_types.h>
#include <hiprand.h>
#include <unistd.h>
#include <stdlib.h>
#include <string.h>
#include <stdio.h>
#include <time.h>
#include <cstdio>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>

#include "support.h"
#include "hash_kernel.cu"
#include "nonce_kernel.cu"
#include "reduction_kernel.cu"

// to activate debug statements
#define DEBUG 1

#define BILLION  1000000000.0

// program constants
#define BLOCK_SIZE 1024
#define SEED       123

// solution constants
#define MAX     123123123
#define TARGET  20

// functions used
void read_file(char* file, unsigned int* transactions, unsigned int n_transactions);
void err_check(hipError_t ret, char* msg, int exit_code);


/* Main ------------------ //
*   This is the main program.
*/
int main(int argc, char* argv[]) {

    // Catch console errors
    if (argc != 6) {
        printf("USE LIKE THIS: gpu_mining transactions.csv n_transactions trials out.csv time.csv\n");
        return EXIT_FAILURE;
    }


    // Output files
    FILE* output_file = fopen(argv[4], "w");
    FILE* time_file = fopen(argv[5], "w");

    // Read in the transactions
    unsigned int n_transactions = strtoul(argv[2], NULL, 10);
    unsigned int* transactions = (unsigned int*)calloc(n_transactions, sizeof(unsigned int));
    read_file(argv[1], transactions, n_transactions);

    // get the number of trials
    unsigned int trials = strtoul(argv[3], NULL, 10);


    // -------- Start Mining ------------------------------------------------------- //
    // ----------------------------------------------------------------------------- //

    // Set timer and cuda error return
    struct timespec start, end;
    clock_gettime(CLOCK_REALTIME, &start);
    hipError_t cuda_ret;

    // To use with kernels
    int num_blocks = ceil((float)trials / (float)BLOCK_SIZE);
    dim3 dimGrid(num_blocks, 1, 1);
    dim3 dimBlock(BLOCK_SIZE, 1, 1);


    // ------ Step 1: generate the nonce values ------ //

    // Allocate the nonce device memory
    unsigned int* device_nonce_array;
    cuda_ret = hipMalloc((void**)&device_nonce_array, trials * sizeof(unsigned int));
    err_check(cuda_ret, (char*)"Unable to allocate nonces to device memory!", 1);

    // Launch the nonce kernel
    nonce_kernel << < dimGrid, dimBlock >> > (
        device_nonce_array, // put nonces into here
        trials,             // size of array
        MAX,                // to mod with
        SEED                // random seed
        );
    cuda_ret = hipDeviceSynchronize();
    err_check(cuda_ret, (char*)"Unable to launch nonce kernel!", 2);

    // Get nonces from device memory
    unsigned int* nonce_array = (unsigned int*)calloc(trials, sizeof(unsigned int));
    cuda_ret = hipMemcpy(nonce_array, device_nonce_array, trials * sizeof(unsigned int), hipMemcpyDeviceToHost);
    err_check(cuda_ret, (char*)"Unable to read nonce from device memory!", 3);


    // ------ Step 2: Generate the hash values ------ //

    // TODO Problem 1: perform this hash generation in the GPU
    // Hint: You need both nonces and transactions to compute a hash.

    // Allocate hash device memory
    unsigned int* device_hash_array;
    cuda_ret = hipMalloc((void**)&device_hash_array, trials * sizeof(unsigned int));
    err_check(cuda_ret, (char*)"Unable to allocate hash to device memory!", 4);

    // Allocate transactions device memory
    unsigned int* device_transactions;
    cuda_ret = hipMalloc((void**)&device_transactions, n_transactions * sizeof(unsigned int));
    err_check(cuda_ret, (char*)"Unable to allocate transactions to device memory!", 5);

    // Copy transactions data to device memory
    cuda_ret = hipMemcpy(device_transactions, transactions, n_transactions * sizeof(unsigned int), hipMemcpyHostToDevice);
    err_check(cuda_ret, (char*)"Unable to read transactions from host memory!", 6);

    // Launch the hash kernel
    hash_kernel << < dimGrid, dimBlock >> > (
        device_hash_array,      // put hash values in here
        device_nonce_array,     // calculated nonce array
        trials,                 // Number of trials
        device_transactions,    // Transactions array
        n_transactions,         // Number of transactions
        MAX                     // to mod with       
        );
    cuda_ret = hipDeviceSynchronize();
    err_check(cuda_ret, (char*)"Unable to launch hash kernel!", 7);

    // Get hash values from device memory
    unsigned int* hash_array = (unsigned int*)calloc(trials, sizeof(unsigned int));
    cuda_ret = hipMemcpy(hash_array, device_hash_array, trials * sizeof(unsigned int), hipMemcpyDeviceToHost);
    err_check(cuda_ret, (char*)"Unable to read hash from device memory!", 8);

    // Free memory
    free(transactions);
    hipFree(device_transactions);

    // ------ Step 3: Find the nonce with the minimum hash value ------ //
    // TODO Problem 2: find the minimum in the GPU by reduction

    unsigned int min_hash = MAX;
    unsigned int min_nonce = MAX;

    // Allocate memory to min hash and min nonce on device
    unsigned int* d_min_hash;
    unsigned int* d_min_nonce;
    hipMalloc(&d_min_hash, sizeof(unsigned int));
    hipMalloc(&d_min_nonce, sizeof(unsigned int));

    // Copy initial min hash and nonce value to device
    hipMemcpy(d_min_hash, &min_hash, sizeof(unsigned int), hipMemcpyHostToDevice);
    hipMemcpy(d_min_nonce, &min_nonce, sizeof(unsigned int), hipMemcpyHostToDevice);

    // Launch the reudction kernel
    reduction_kernel << < dimGrid, dimBlock >> > (
        device_hash_array,      // device hash array
        device_nonce_array,     // device nonce array
        trials,                 // number of trials
        MAX,                    // MAX constant value
        d_min_hash,             // for storing device min hash
        d_min_nonce             // for storing device min nonce       
        );
    hipDeviceSynchronize();

    // Copy min hash and nonce value from device
    hipMemcpy(&min_hash, d_min_hash, sizeof(unsigned int), hipMemcpyDeviceToHost);
    hipMemcpy(&min_nonce, d_min_nonce, sizeof(unsigned int), hipMemcpyDeviceToHost);

    // Free memory
    free(nonce_array);
    free(hash_array);
    hipFree(device_hash_array);
    hipFree(device_nonce_array);
    hipFree(d_min_hash);   
    hipFree(d_min_nonce);

    clock_gettime(CLOCK_REALTIME, &end);

    // Calculate elapsed time
    double ElapsedTime = (end.tv_sec - start.tv_sec) +
        (end.tv_nsec - start.tv_nsec) / BILLION;

    // ----------------------------------------------------------------------------- //
    // -------- Finish Mining ------------------------------------------------------ //


    // Get if suceeded
    char* res = (char*)malloc(8 * sizeof(char));
    if (min_hash < TARGET)  res = (char*)"Success!";
    else                    res = (char*)"Failure.";

    // Show results in console
    if (DEBUG)
        printf("%s\n   Min hash:  %u\n   Min nonce: %u\n  %.2f seconds\n",
            res,
            min_hash,
            min_nonce,
            ElapsedTime
        );

    // Print results
    fprintf(output_file, "%s\n%u\n%u\n", res, min_hash, min_nonce);
    fprintf(time_file, "%.2f\n", ElapsedTime);

    // Cleanup
    fclose(time_file);
    fclose(output_file);

    return 0;
} // End Main -------------------------------------------- //


/* Read File -------------------- //
*   Reads in a file of transactions.
*/
void read_file(char* file, unsigned int* transactions, unsigned int n_transactions) {

    // open file
    FILE* trans_file = fopen(file, "r");
    if (trans_file == NULL)
        fprintf(stderr, "ERROR: could not read the transaction file.\n"),
        exit(-1);

    // read items
    char line[100] = { 0 };
    for (int i = 0; i < n_transactions && fgets(line, 100, trans_file); ++i) {
        char* p;
        transactions[i] = strtof(line, &p);
    }

    fclose(trans_file);

} // End Read File ------------- //



/* Error Check ----------------- //
*   Exits if there is a CUDA error.
*/
void err_check(hipError_t ret, char* msg, int exit_code) {
    if (ret != hipSuccess)
        fprintf(stderr, "%s \"%s\".\n", msg, hipGetErrorString(ret)),
        exit(exit_code);
} // End Error Check ----------- //
